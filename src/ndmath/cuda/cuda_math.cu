#include "hip/hip_runtime.h"
#include "cuda_math.h"
#include <hip/hip_runtime.h>
#include "../../ndarray.h"
#include "../../initializers.h"
#include "../../debug.h"
#include <float.h>
#include <hipsolver.h>

#define CHECK_CUDA(func) do { \
  hipError_t status = (func); \
  if (status != hipSuccess) { \
    printf("CUDA API failed at line %d with error: %s\n", \
           __LINE__, hipGetErrorString(status)); \
    return EXIT_FAILURE; \
  } \
} while (0)

#define CHECK_CUSOLVER(func) do { \
  hipsolverStatus_t status = (func); \
  if (status != HIPSOLVER_STATUS_SUCCESS) { \
    printf("cuSOLVER API failed at line %d with error: %d\n", \
           __LINE__, status); \
    return EXIT_FAILURE; \
  } \
} while (0)

#define CHECK_CUBLAS(func) do { \
  hipsolverStatus_t status = (func); \
  if (status != HIPSOLVER_STATUS_SUCCESS) { \
    printf("cuBLAS API failed at line %d with error: %d\n", \
           __LINE__, status); \
    return EXIT_FAILURE; \
  } \
} while (0)

__device__
float sinc(float number) {
    if (number == 0.0) {
        return 1.0;
    } else {
        return sinf(M_PI * number) / (M_PI * number);
    }
}

__global__
void sincFloatKernel(float* d_array, int size) {
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    if (index < size) {

        d_array[index] = sinc(d_array[index]);
    }
}

__global__
void truncFloatKernel(float* d_array, int size) {
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    if (index < size) {
        d_array[index] = truncf(d_array[index]);
    }
}

__device__
int roundFloatToNearestInt(float number) {
    float rounded = rintf(number);
    int floorInt = (int)floorf(number);

    // Check if the rounded value is halfway between two integers
    if (rounded - floorInt == 0.5 && ((int)rounded % 2 != 0)) {
        rounded -= 1.0;
    }

    return (int)rounded;
}

__global__
void rintFloatKernel(float* d_array, int size) {
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    if (index < size) {
        d_array[index] = roundFloatToNearestInt(d_array[index]);
    }
}

__global__
void fixFloatKernel(float* d_array, int size) {
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    if (index < size) {
        d_array[index] = truncf(d_array[index]);
    }
}

__global__
void ceilFloatKernel(float* d_array, int size) {
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    if (index < size) {
        d_array[index] = ceilf(d_array[index]);
    }
}

__global__
void floorFloatKernel(float* d_array, int size) {
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    if (index < size) {
        d_array[index] = floorf(d_array[index]);
    }
}

__global__
void arcsinhFloatKernel(float* d_array, int size) {
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    if (index < size) {
        d_array[index] = asinhf(d_array[index]);
    }
}

__global__
void arccoshFloatKernel(float* d_array, int size) {
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    if (index < size) {
        d_array[index] = acoshf(d_array[index]);
    }
}

__global__
void arctanhFloatKernel(float* d_array, int size) {
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    if (index < size) {
        d_array[index] = atanhf(d_array[index]);
    }
}

__global__
void sinhFloatKernel(float* d_array, int size) {
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    if (index < size) {
        d_array[index] = sinhf(d_array[index]);
    }
}

__global__
void coshFloatKernel(float* d_array, int size) {
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    if (index < size) {
        d_array[index] = coshf(d_array[index]);
    }
}

__global__
void tanhFloatKernel(float* d_array, int size) {
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    if (index < size) {
        d_array[index] = tanhf(d_array[index]);
    }
}

__global__
void degreesFloatKernel(float* d_array, int size) {
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    if (index < size) {
        d_array[index] = d_array[index] * (180.0 / 3.1415926535);
    }
}

__global__
void radiansFloatKernel(float* d_array, int size) {
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    if (index < size) {
        d_array[index] = d_array[index] * (3.1415926535 / 180.0);
    }
}

__global__
void arcsinFloatKernel(float* d_array, int size) {
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    if (index < size) {
        d_array[index] = asinf(d_array[index]);
    }
}

__global__
void arccosFloatKernel(float* d_array, int size) {
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    if (index < size) {
        d_array[index] = acosf(d_array[index]);
    }
}

__global__
void arctanFloatKernel(float* d_array, int size) {
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    if (index < size) {
        d_array[index] = atanf(d_array[index]);
    }
}

__global__
void absFloatKernel(float* d_array, int size) {
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    if (index < size) {
        d_array[index] = fabsf(d_array[index]);
    }
}

__global__
void sinFloatKernel(float* d_array, int size) {
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    if (index < size) {
        d_array[index] = sinf(d_array[index]);
    }
}

__global__
void cosFloatKernel(float* d_array, int size) {
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    if (index < size) {
        d_array[index] = cosf(d_array[index]);
    }
}

__global__
void tanFloatKernel(float* d_array, int size) {
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    if (index < size) {
        d_array[index] = tanf(d_array[index]);
    }
}

__global__
void expm1FloatKernel(float* d_array, int size) {
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    if (index < size) {
        d_array[index] = expm1f(d_array[index]);
    }
}

__global__
void expFloatKernel(float* d_array, int size) {
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    if (index < size) {
        d_array[index] = expf(d_array[index]);
    }
}

__global__
void sqrtFloatKernel(float* d_array, int size) {
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    if (index < size) {
        d_array[index] = sqrtf(d_array[index]);
    }
}

__global__
void logFloatKernel(float* d_array, int size) {
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    if (index < size) {
        d_array[index] = logf(d_array[index]);
    }
}

__global__
void logbFloatKernel(float* d_array, int size) {
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    if (index < size) {
        d_array[index] = logbf(d_array[index]);
    }
}

__global__
void log2FloatKernel(float* d_array, int size) {
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    if (index < size) {
        d_array[index] = log2f(d_array[index]);
    }
}

__global__
void log1pFloatKernel(float* d_array, int size) {
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    if (index < size) {
        d_array[index] = log1pf(d_array[index]);
    }
}

__global__
void log10FloatKernel(float* d_array, int size) {
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    if (index < size) {
        d_array[index] = log10f(d_array[index]);
    }
}

__global__ void
add_vectors_float_kernel(float *a, float *b, float *result, int size) {
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    if (index < size) {
        result[index] = a[index] + b[index];
    }
}

__global__ void
subtract_vectors_float_kernel(float *a, float *b, float *result, int size) {
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    if (index < size) {
        result[index] = a[index] - b[index];
    }
}

__global__ void
divide_vectors_float_kernel(float *a, float *b, float *result, int size) {
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    if (index < size) {
        result[index] = a[index] / b[index];
    }
}

__global__ void
multiply_vectors_float_kernel(float *a, float *b, float *result, int size) {
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    if (index < size) {
        result[index] = a[index] * b[index];
    }
}

__global__ void
fmodf_float_kernel(float *a, float *b, float *result, int size) {
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    if (index < size) {
        result[index] = fmodf(a[index], b[index]);
    }
}

__global__ void
pow_float_kernel(float *a, float *b, float *result, int size) {
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    if (index < size) {
        result[index] = powf(a[index], b[index]);
    }
}

__global__ void
max_reduce_naive(float * d_out, float * d_in, int n) {
    extern __shared__ float sdata[];

    int myId = threadIdx.x + blockDim.x * blockIdx.x;
    int tid = threadIdx.x;

    // load shared mem from global mem
    sdata[tid] = d_in[myId];
    __syncthreads();            // make sure entire block is loaded!

    // do reduction in shared mem
    for (unsigned int s = blockDim.x / 2; s > 0; s >>= 1) {
        if (tid < s) {
            sdata[tid] = fmaxf(sdata[tid], sdata[tid + s]);
        }
        __syncthreads();        // make sure all adds at one stage are done!
    }

    // only thread 0 writes result for this block back to global mem
    if (tid == 0) {
        d_out[blockIdx.x] = sdata[0];
    }
}

__global__ void
min_reduce_naive(float * d_out, float * d_in, int n) {
    extern __shared__ float sdata[];

    int myId = threadIdx.x + blockDim.x * blockIdx.x;
    int tid = threadIdx.x;

    // load shared mem from global mem
    sdata[tid] = d_in[myId];
    __syncthreads();            // make sure entire block is loaded!

    // do reduction in shared mem
    for (unsigned int s = blockDim.x / 2; s > 0; s >>= 1) {
        if (tid < s) {
            sdata[tid] = fminf(sdata[tid], sdata[tid + s]);
        }
        __syncthreads();        // make sure all adds at one stage are done!
    }

    // only thread 0 writes result for this block back to global mem
    if (tid == 0) {
        d_out[blockIdx.x] = sdata[0];
    }
}

__global__
void array_equals_float(float *a, float *b, int *result, int n) {
    int idx = threadIdx.x + blockDim.x * blockIdx.x;
    if (idx < n) {
        if (a[idx] != b[idx]) {
            atomicExch(result, 0); // If any element is not equal, set 'equal' to 0
        }
    }
}

__global__
void array_sum_float(float *a, float *result, int n) {
    extern __shared__ float sdata[];

    // each thread loads one element from global to shared mem
    unsigned int tid = threadIdx.x;
    unsigned int i = blockIdx.x * (blockDim.x * 2) + threadIdx.x;

    float x = 0;
    if (i < n) x += a[i];
    if (i + blockDim.x < n) x += a[i + blockDim.x];
    sdata[tid] = x;
    __syncthreads();

    // do reduction in shared mem
    for (unsigned int s=blockDim.x/2; s>0; s>>=1) {
        if (tid < s) {
            sdata[tid] += sdata[tid + s];
        }
        __syncthreads();
    }

    // write result for this block to global mem
    if (tid == 0) atomicAdd(result, sdata[0]);
}

// CUDA Kernel for Matrix Multiplication for non-square matrices
__global__ void
matmul_float_kernel(float* A, float* B, float* C, int widthA, int heightA, int widthB) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < heightA && col < widthB) {
        float sum = 0;
        for(int i = 0; i < widthA; ++i) {
            sum += A[row * widthA + i] * B[i * widthB + col];
        }
        C[row * widthB + col] = sum;
    }
}

__global__
void fill_float_kernel(float* array, int n, float value) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if(idx < n) {
        array[idx] = value;
    }
}

extern "C" {

    int
    cuda_det_float(float *a, float *result, int n) {
        int N = n;
        float *d_A = a;
        hipsolverHandle_t cusolverH = NULL;
        hipStream_t stream = NULL;
        hipblasHandle_t cublasH = NULL;
        hipsolverStatus_t cusolver_status = HIPSOLVER_STATUS_SUCCESS;

        CHECK_CUSOLVER(hipsolverDnCreate(&cusolverH));
        CHECK_CUDA(hipStreamCreateWithFlags(&stream, hipStreamNonBlocking));
        CHECK_CUSOLVER(hipsolverSetStream(cusolverH, stream));
        hipblasCreate(&cublasH);
        hipblasSetStream(cublasH, stream);

        int* d_Ipiv; // pivot array
        int* d_info;  // info on success or failure
        float* d_U; // U matrix of LU decomposition

        CHECK_CUDA(hipMalloc(&d_Ipiv, N*sizeof(int)));
        CHECK_CUDA(hipMalloc(&d_info, sizeof(int)));
        CHECK_CUDA(hipMalloc(&d_U, N*N*sizeof(float)));

        // copy A to U as hipsolverDnSgetrf works in place
        CHECK_CUDA(hipMemcpy(d_U, d_A, N*N*sizeof(float), hipMemcpyDeviceToDevice));

        // LU decompose
        cusolver_status = hipsolverDnSgetrf(cusolverH, N, N, d_U, N, NULL, d_Ipiv, d_info);
        if (cusolver_status != HIPSOLVER_STATUS_SUCCESS) {
            // handle error
            printf("LU decomposition failed\n");
            exit(1);
        }

        // Find determinant by product of diagonal elements
        float det = 1.0f;
        for (int i = 0; i < N; i++) {
            float elem;
            CHECK_CUDA(hipMemcpy(&elem, d_U + i * N + i, sizeof(float), hipMemcpyDeviceToHost));
            // Check for potential overflow
            if (fabsf(elem) > FLT_MAX / fabsf(det)) {
                // Handle overflow here, e.g., return a special value or throw an error
                printf("Overflow detected in det\n");
                exit(1);
            }
            if (!isnan(elem) && !isinf(elem)) {
                det *= elem;
            }
        }

        // Analyze pivot array to calculate number of permutations
        int* h_Ipiv = new int[N];
        CHECK_CUDA(hipMemcpy(h_Ipiv, d_Ipiv, N*sizeof(int), hipMemcpyDeviceToHost));

        int numPermutations = 0;
        for(int i = 0; i < N; i++) {
            if(i+1 != h_Ipiv[i]) numPermutations++;
        }

        if(numPermutations % 2 != 0) det = -det;

        // Cleanup
        if (d_U) hipFree(d_U);
        if (d_Ipiv) hipFree(d_Ipiv);
        if (d_info) hipFree(d_info);
        if (cublasH) hipblasDestroy(cublasH);
        if (cusolverH) hipsolverDnDestroy(cusolverH);
        if (stream) hipStreamDestroy(stream);

        CHECK_CUDA(hipMemcpy(result, &det, sizeof(float), hipMemcpyHostToDevice));
        return 1;
    }

    void
    cuda_fill_float(float *a, float value, int n) {
        int blockSize = 256;
        int gridSize = (n + blockSize - 1) / blockSize;

        fill_float_kernel<<<gridSize, blockSize>>>(a, n, value);
        hipDeviceSynchronize();
    }

    void
    cuda_matmul_float(int nblocks, float *a, float *b, float *rtn, int widthA, int heightA, int widthB) {
        dim3 blockSize(16, 16); // Use a block size appropriate for your hardware
        dim3 gridSize((widthB + blockSize.x - 1) / blockSize.x, (heightA + blockSize.y - 1) / blockSize.y);

        matmul_float_kernel<<<gridSize, blockSize>>>(a, b, rtn, widthA, heightA, widthB);
        hipDeviceSynchronize();
    }

    void
    cuda_sum_float(int nblocks, float *a, float *rtn, int nelements) {
        float *d_sum;
        int blockSize = 256;  // Number of threads per block. This is a typical choice.
        int numBlocks = (nblocks + blockSize * 2 - 1) / (blockSize * 2);  // Number of blocks in the grid.
        hipMalloc((void **) &d_sum, sizeof(float));

        hipMemcpy(d_sum, rtn, sizeof(float), hipMemcpyHostToDevice);
        array_sum_float<<<numBlocks, blockSize, blockSize * sizeof(float)>>>(a, d_sum, nelements);
        hipMemcpy(rtn, d_sum, sizeof(float), hipMemcpyDeviceToHost);
        hipDeviceSynchronize();
    }

    int
    cuda_svd_float(float *d_A, float *d_U, float *d_V, float *d_S, int m, int n) {
        hipsolverHandle_t cusolverH = NULL;  // cuSOLVER handle
        hipStream_t stream = NULL;  // CUDA stream
        hipsolverGesvdjInfo_t gesvdj_params = NULL;  // configuration of gesvdj
        CHECK_CUSOLVER(hipsolverDnCreate(&cusolverH));
        CHECK_CUDA(hipStreamCreateWithFlags(&stream, hipStreamNonBlocking));
        CHECK_CUSOLVER(hipsolverSetStream(cusolverH, stream));
        CHECK_CUSOLVER(hipsolverDnCreateGesvdjInfo(&gesvdj_params));

        // Set desired configuration of gesvdj
        CHECK_CUSOLVER(hipsolverDnXgesvdjSetTolerance(
                gesvdj_params,
                1.e-7));
        CHECK_CUSOLVER(hipsolverDnXgesvdjSetMaxSweeps(
                gesvdj_params,
                15));

        // Perform SVD
        // Note: This is just a skeleton code. Please handle CUDA errors appropriately
        int* devInfo = NULL;  // info on gesvdj convergence
        CHECK_CUDA(hipMalloc((void**)&devInfo, sizeof(int)));
        int lwork = 0;
        float *d_work = NULL;
        CHECK_CUSOLVER(hipsolverDnSgesvdj_bufferSize(
                cusolverH,
                HIPSOLVER_EIG_MODE_VECTOR,  // compute eigenvectors
                0,  // number of singular values to compute, 0 for all
                m,
                n,
                d_A,
                m,  // leading dimension of A
                d_S,
                d_U,
                m,  // leading dimension of U
                d_V,
                n,  // leading dimension of V
                &lwork,
                gesvdj_params));

        CHECK_CUDA(hipMalloc((void**)&d_work , sizeof(float) * lwork));
        CHECK_CUSOLVER(hipsolverDnSgesvdj(
                cusolverH,
                HIPSOLVER_EIG_MODE_VECTOR,  // compute eigenvectors
                0,  // number of singular values to compute, 0 for all
                m,
                n,
                d_A,
                m,  // leading dimension of A
                d_S,
                d_U,
                m,  // leading dimension of U
                d_V,
                n,  // leading dimension of V
                d_work,
                lwork,
                devInfo,
                gesvdj_params));

        // Synchronize to ensure computation is finished
        CHECK_CUDA(hipDeviceSynchronize());
        if (devInfo) CHECK_CUDA(hipFree(devInfo));
        if (cusolverH) CHECK_CUSOLVER(hipsolverDnDestroy(cusolverH));
        if (stream) CHECK_CUDA(hipStreamDestroy(stream));
        if (gesvdj_params) CHECK_CUSOLVER(hipsolverDnDestroyGesvdjInfo(gesvdj_params));

        return 1;
    }

    float
    cuda_max_float(float *a, int nelements) {
        int size = nelements;
        float *d_out;
        int blockSize = 256;  // Number of threads per block. This is a typical choice.

        int current_size = size;
        float *d_current_in = a;
        while(current_size > 1) {
            int blocks = (current_size + blockSize - 1) / blockSize;
            hipMalloc((void **) &d_out, blocks * sizeof(float));
            max_reduce_naive<<<blocks, blockSize, blockSize * sizeof(float)>>>(d_out, d_current_in, current_size);

            if (d_current_in != a) { // Free the intermediate input arrays
                hipFree(d_current_in);
            }

            // Prepare for the next iteration
            d_current_in = d_out;
            current_size = blocks;
        }
        hipDeviceSynchronize();

        // copy the result back to the host
        float max_value;
        hipMemcpy(&max_value, d_out, sizeof(float), hipMemcpyDeviceToHost);

        return max_value;
    }

    float
    cuda_min_float(float *a, int nelements) {
        int size = nelements;
        float *d_out;
        int blockSize = 256;  // Number of threads per block. This is a typical choice.

        int current_size = size;
        float *d_current_in = a;
        while(current_size > 1) {
            int blocks = (current_size + blockSize - 1) / blockSize;
            hipMalloc((void **) &d_out, blocks * sizeof(float));
            min_reduce_naive<<<blocks, blockSize, blockSize * sizeof(float)>>>(d_out, d_current_in, current_size);

            if (d_current_in != a) { // Free the intermediate input arrays
                hipFree(d_current_in);
            }

            // Prepare for the next iteration
            d_current_in = d_out;
            current_size = blocks;
        }
        hipDeviceSynchronize();

        // copy the result back to the host
        float min_value;
        hipMemcpy(&min_value, d_out, sizeof(float), hipMemcpyDeviceToHost);

        return min_value;
    }

    int
    cuda_equal_float(int nblocks, float *a, float *b, int nelements) {
        int blockSize = 256;  // Number of threads per block. This is a typical choice.
        int result = 1;
        int *d_equal;
        // Allocate GPU memory for the result
        hipMalloc(&d_equal, sizeof(int));
        hipMemcpy(d_equal, &result, sizeof(int), hipMemcpyHostToDevice);
        int numBlocks = (nblocks + blockSize - 1) / blockSize;  // Number of blocks in the grid.
        array_equals_float<<<numBlocks, blockSize>>>(a, b, d_equal, nelements);
        hipDeviceSynchronize();
        hipMemcpy(&result, d_equal, sizeof(int), hipMemcpyDeviceToHost);
        hipFree(d_equal);
        return result;
    }

    void
    cuda_pow_float(int nblocks, float *a, float *b, float *rtn, int nelements) {
        int blockSize = 256;  // Number of threads per block. This is a typical choice.
        int numBlocks = (nblocks + blockSize - 1) / blockSize;  // Number of blocks in the grid.
        pow_float_kernel<<<numBlocks, blockSize>>>(a, b, rtn, nelements);
        hipDeviceSynchronize();
    }

    void
    cuda_mod_float(int nblocks, float *a, float *b, float *rtn, int nelements) {
        int blockSize = 256;  // Number of threads per block. This is a typical choice.
        int numBlocks = (nblocks + blockSize - 1) / blockSize;  // Number of blocks in the grid.
        fmodf_float_kernel<<<numBlocks, blockSize>>>(a, b, rtn, nelements);
        hipDeviceSynchronize();
    }

    void
    cuda_multiply_float(int nblocks, float *a, float *b, float *rtn, int nelements) {
        int blockSize = 256;  // Number of threads per block. This is a typical choice.
        int numBlocks = (nblocks + blockSize - 1) / blockSize;  // Number of blocks in the grid.
        multiply_vectors_float_kernel<<<numBlocks, blockSize>>>(a, b, rtn, nelements);
        hipDeviceSynchronize();
    }

    void
    cuda_divide_float(int nblocks, float *a, float *b, float *rtn, int nelements) {
        int blockSize = 256;  // Number of threads per block. This is a typical choice.
        int numBlocks = (nblocks + blockSize - 1) / blockSize;  // Number of blocks in the grid.
        divide_vectors_float_kernel<<<numBlocks, blockSize>>>(a, b, rtn, nelements);
        hipDeviceSynchronize();
    }

    void
    cuda_subtract_float(int nblocks, float *a, float *b, float *rtn, int nelements) {
        int blockSize = 256;  // Number of threads per block. This is a typical choice.
        int numBlocks = (nblocks + blockSize - 1) / blockSize;  // Number of blocks in the grid.
        subtract_vectors_float_kernel<<<numBlocks, blockSize>>>(a, b, rtn, nelements);
        hipDeviceSynchronize();
    }

    void
    cuda_add_float(int nblocks, float *a, float *b, float *rtn, int nelements) {
        int blockSize = 256;  // Number of threads per block. This is a typical choice.
        int numBlocks = (nblocks + blockSize - 1) / blockSize;  // Number of blocks in the grid.
        add_vectors_float_kernel<<<numBlocks, blockSize>>>(a, b, rtn, nelements);
        hipDeviceSynchronize();
    }

    void
    cuda_float_log(int nblocks, float *d_array) {
        int blockSize = 256;  // Number of threads per block. This is a typical choice.
        int numBlocks = (nblocks + blockSize - 1) / blockSize;  // Number of blocks in the grid.
        logFloatKernel<<<numBlocks, blockSize>>>(d_array, nblocks);
        hipDeviceSynchronize();
    }

    void
    cuda_float_logb(int nblocks, float *d_array) {
        int blockSize = 256;  // Number of threads per block. This is a typical choice.
        int numBlocks = (nblocks + blockSize - 1) / blockSize;  // Number of blocks in the grid.
        logbFloatKernel<<<numBlocks, blockSize>>>(d_array, nblocks);
        hipDeviceSynchronize();
    }

    void
    cuda_float_log2(int nblocks, float *d_array) {
        int blockSize = 256;  // Number of threads per block. This is a typical choice.
        int numBlocks = (nblocks + blockSize - 1) / blockSize;  // Number of blocks in the grid.
        log2FloatKernel<<<numBlocks, blockSize>>>(d_array, nblocks);
        hipDeviceSynchronize();
    }

    void
    cuda_float_log1p(int nblocks, float *d_array) {
        int blockSize = 256;  // Number of threads per block. This is a typical choice.
        int numBlocks = (nblocks + blockSize - 1) / blockSize;  // Number of blocks in the grid.
        log1pFloatKernel<<<numBlocks, blockSize>>>(d_array, nblocks);
        hipDeviceSynchronize();
    }

    void
    cuda_float_log10(int nblocks, float *d_array) {
        int blockSize = 256;  // Number of threads per block. This is a typical choice.
        int numBlocks = (nblocks + blockSize - 1) / blockSize;  // Number of blocks in the grid.
        log10FloatKernel<<<numBlocks, blockSize>>>(d_array, nblocks);
        hipDeviceSynchronize();
    }

    void
    cuda_float_sqrt(int nblocks, float *d_array) {
        int blockSize = 256;  // Number of threads per block. This is a typical choice.
        int numBlocks = (nblocks + blockSize - 1) / blockSize;  // Number of blocks in the grid.
        sqrtFloatKernel<<<numBlocks, blockSize>>>(d_array, nblocks);
        hipDeviceSynchronize();
    }

    void
    cuda_float_exp(int nblocks, float *d_array) {
        int blockSize = 256;  // Number of threads per block. This is a typical choice.
        int numBlocks = (nblocks + blockSize - 1) / blockSize;  // Number of blocks in the grid.
        expFloatKernel<<<numBlocks, blockSize>>>(d_array, nblocks);
        hipDeviceSynchronize();
    }

    void
    cuda_float_abs(int nblocks, float *d_array) {
        int blockSize = 256;  // Number of threads per block. This is a typical choice.
        int numBlocks = (nblocks + blockSize - 1) / blockSize;  // Number of blocks in the grid.
        absFloatKernel<<<numBlocks, blockSize>>>(d_array, nblocks);
        hipDeviceSynchronize();
    }

    void
    cuda_float_expm1(int nblocks, float *d_array) {
        int blockSize = 256;  // Number of threads per block. This is a typical choice.
        int numBlocks = (nblocks + blockSize - 1) / blockSize;  // Number of blocks in the grid.
        expm1FloatKernel<<<numBlocks, blockSize>>>(d_array, nblocks);
        hipDeviceSynchronize();
    }

    void
    cuda_float_sin(int nblocks, float *d_array) {
        int blockSize = 256;  // Number of threads per block. This is a typical choice.
        int numBlocks = (nblocks + blockSize - 1) / blockSize;  // Number of blocks in the grid.
        sinFloatKernel<<<numBlocks, blockSize>>>(d_array, nblocks);
        hipDeviceSynchronize();
    }

    void
    cuda_float_cos(int nblocks, float *d_array) {
        int blockSize = 256;  // Number of threads per block. This is a typical choice.
        int numBlocks = (nblocks + blockSize - 1) / blockSize;  // Number of blocks in the grid.
        cosFloatKernel<<<numBlocks, blockSize>>>(d_array, nblocks);
        hipDeviceSynchronize();
    }

    void
    cuda_float_tan(int nblocks, float *d_array) {
        int blockSize = 256;  // Number of threads per block. This is a typical choice.
        int numBlocks = (nblocks + blockSize - 1) / blockSize;  // Number of blocks in the grid.
        tanFloatKernel<<<numBlocks, blockSize>>>(d_array, nblocks);
        hipDeviceSynchronize();
    }

    void
    cuda_float_arcsin(int nblocks, float *d_array) {
        int blockSize = 256;  // Number of threads per block. This is a typical choice.
        int numBlocks = (nblocks + blockSize - 1) / blockSize;  // Number of blocks in the grid.
        arcsinFloatKernel<<<numBlocks, blockSize>>>(d_array, nblocks);
        hipDeviceSynchronize();
    }

    void
    cuda_float_arctan(int nblocks, float *d_array) {
        int blockSize = 256;  // Number of threads per block. This is a typical choice.
        int numBlocks = (nblocks + blockSize - 1) / blockSize;  // Number of blocks in the grid.
        arctanFloatKernel<<<numBlocks, blockSize>>>(d_array, nblocks);
        hipDeviceSynchronize();
    }

    void
    cuda_float_arccos(int nblocks, float *d_array) {
        int blockSize = 256;  // Number of threads per block. This is a typical choice.
        int numBlocks = (nblocks + blockSize - 1) / blockSize;  // Number of blocks in the grid.
        arccosFloatKernel<<<numBlocks, blockSize>>>(d_array, nblocks);
        hipDeviceSynchronize();
    }

    void
    cuda_float_radians(int nblocks, float *d_array) {
        int blockSize = 256;  // Number of threads per block. This is a typical choice.
        int numBlocks = (nblocks + blockSize - 1) / blockSize;  // Number of blocks in the grid.
        radiansFloatKernel<<<numBlocks, blockSize>>>(d_array, nblocks);
        hipDeviceSynchronize();
    }

    void
    cuda_float_degrees(int nblocks, float *d_array) {
        int blockSize = 256;  // Number of threads per block. This is a typical choice.
        int numBlocks = (nblocks + blockSize - 1) / blockSize;  // Number of blocks in the grid.
        degreesFloatKernel<<<numBlocks, blockSize>>>(d_array, nblocks);
        hipDeviceSynchronize();
    }

    void
    cuda_float_sinh(int nblocks, float *d_array) {
        int blockSize = 256;  // Number of threads per block. This is a typical choice.
        int numBlocks = (nblocks + blockSize - 1) / blockSize;  // Number of blocks in the grid.
        sinhFloatKernel<<<numBlocks, blockSize>>>(d_array, nblocks);
        hipDeviceSynchronize();
    }

    void
    cuda_float_cosh(int nblocks, float *d_array) {
        int blockSize = 256;  // Number of threads per block. This is a typical choice.
        int numBlocks = (nblocks + blockSize - 1) / blockSize;  // Number of blocks in the grid.
        coshFloatKernel<<<numBlocks, blockSize>>>(d_array, nblocks);
        hipDeviceSynchronize();
    }

    void
    cuda_float_tanh(int nblocks, float *d_array) {
        int blockSize = 256;  // Number of threads per block. This is a typical choice.
        int numBlocks = (nblocks + blockSize - 1) / blockSize;  // Number of blocks in the grid.
        tanhFloatKernel<<<numBlocks, blockSize>>>(d_array, nblocks);
        hipDeviceSynchronize();
    }

    void
    cuda_float_arcsinh(int nblocks, float *d_array) {
        int blockSize = 256;  // Number of threads per block. This is a typical choice.
        int numBlocks = (nblocks + blockSize - 1) / blockSize;  // Number of blocks in the grid.
        arcsinhFloatKernel<<<numBlocks, blockSize>>>(d_array, nblocks);
        hipDeviceSynchronize();
    }

    void
    cuda_float_arccosh(int nblocks, float *d_array) {
        int blockSize = 256;  // Number of threads per block. This is a typical choice.
        int numBlocks = (nblocks + blockSize - 1) / blockSize;  // Number of blocks in the grid.
        arccoshFloatKernel<<<numBlocks, blockSize>>>(d_array, nblocks);
        hipDeviceSynchronize();
    }

    void
    cuda_float_arctanh(int nblocks, float *d_array) {
        int blockSize = 256;  // Number of threads per block. This is a typical choice.
        int numBlocks = (nblocks + blockSize - 1) / blockSize;  // Number of blocks in the grid.
        arctanhFloatKernel<<<numBlocks, blockSize>>>(d_array, nblocks);
        hipDeviceSynchronize();
    }

    void
    cuda_float_rint(int nblocks, float *d_array) {
        int blockSize = 256;  // Number of threads per block. This is a typical choice.
        int numBlocks = (nblocks + blockSize - 1) / blockSize;  // Number of blocks in the grid.
        rintFloatKernel<<<numBlocks, blockSize>>>(d_array, nblocks);
        hipDeviceSynchronize();
    }

    void
    cuda_float_fix(int nblocks, float *d_array) {
        int blockSize = 256;  // Number of threads per block. This is a typical choice.
        int numBlocks = (nblocks + blockSize - 1) / blockSize;  // Number of blocks in the grid.
        fixFloatKernel<<<numBlocks, blockSize>>>(d_array, nblocks);
        hipDeviceSynchronize();
    }

    void
    cuda_float_ceil(int nblocks, float *d_array) {
        int blockSize = 256;  // Number of threads per block. This is a typical choice.
        int numBlocks = (nblocks + blockSize - 1) / blockSize;  // Number of blocks in the grid.
        ceilFloatKernel<<<numBlocks, blockSize>>>(d_array, nblocks);
        hipDeviceSynchronize();
    }

    void
    cuda_float_floor(int nblocks, float *d_array) {
        int blockSize = 256;  // Number of threads per block. This is a typical choice.
        int numBlocks = (nblocks + blockSize - 1) / blockSize;  // Number of blocks in the grid.
        floorFloatKernel<<<numBlocks, blockSize>>>(d_array, nblocks);
        hipDeviceSynchronize();
    }

    void
    cuda_float_trunc(int nblocks, float *d_array) {
        int blockSize = 256;  // Number of threads per block. This is a typical choice.
        int numBlocks = (nblocks + blockSize - 1) / blockSize;  // Number of blocks in the grid.
        truncFloatKernel<<<numBlocks, blockSize>>>(d_array, nblocks);
        hipDeviceSynchronize();
    }

    void
    cuda_float_sinc(int nblocks, float *d_array) {
        int blockSize = 256;  // Number of threads per block. This is a typical choice.
        int numBlocks = (nblocks + blockSize - 1) / blockSize;  // Number of blocks in the grid.
        sincFloatKernel<<<numBlocks, blockSize>>>(d_array, nblocks);
        hipDeviceSynchronize();
    }

    NDArray*
    NDArrayMathGPU_ElementWise(NDArray* ndarray, ElementWiseFloatGPUOperation op) {
        NDArray *rtn = NDArray_Copy(ndarray, NDArray_DEVICE(ndarray));
        op(NDArray_NUMELEMENTS(rtn), NDArray_FDATA(rtn));
        return rtn;
    }

}