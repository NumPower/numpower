#include "../../../config.h"
#include "cuda_dnn.cuh"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <vector>
#include <stdio.h>

#ifdef HAVE_CUDNN

#include <hipDNN.h>

#define checkCUDNN(expression)                               \
  {                                                          \
    hipdnnStatus_t status = (expression);                     \
    if (status != HIPDNN_STATUS_SUCCESS) {                    \
      printf("\nError on line %d (%d)", __LINE__, status);     \
    }                                                        \
  }

float*
cuda_dnn_conv2d_float32(float *input, int num_channels, int num_elements, int batch_size, int height, int width,
                        int *output_shape, int kernel_size, char padding) {
    hipdnnHandle_t cudnn;
    hipdnnCreate(&cudnn);

    hipdnnTensorDescriptor_t input_descriptor;
    checkCUDNN(hipdnnCreateTensorDescriptor(&input_descriptor));
    checkCUDNN(hipdnnSetTensor4dDescriptor(input_descriptor,
            /*format=*/HIPDNN_TENSOR_NHWC,
            /*dataType=*/HIPDNN_DATA_FLOAT,
            /*batch_size=*/batch_size,
            /*channels=*/num_channels,
            /*image_height=*/height,
            /*image_width=*/width));

    hipdnnFilterDescriptor_t kernel_descriptor;
    checkCUDNN(hipdnnCreateFilterDescriptor(&kernel_descriptor));
    checkCUDNN(hipdnnSetFilter4dDescriptor(kernel_descriptor,
            /*dataType=*/HIPDNN_DATA_FLOAT,
            /*format=*/HIPDNN_TENSOR_NCHW,
            /*out_channels=*/kernel_size,
            /*in_channels=*/num_channels,
            /*kernel_height=*/3,
            /*kernel_width=*/3));

    int pad_height;
    int pad_width;
    switch (padding) {
        case 'v':
            pad_height = 0;
            pad_width = 0;
            break;
        case 's':
            pad_height = 1;
            pad_width = 1;
            break;
        default:
            pad_height = 0;
            pad_width = 0;
    }

    hipdnnConvolutionDescriptor_t convolution_descriptor;
    checkCUDNN(hipdnnCreateConvolutionDescriptor(&convolution_descriptor));
    checkCUDNN(hipdnnSetConvolution2dDescriptor(convolution_descriptor,
            /*pad_height=*/pad_height,
            /*pad_width=*/pad_width,
            /*vertical_stride=*/1,
            /*horizontal_stride=*/1,
            /*dilation_height=*/1,
            /*dilation_width=*/1,
            /*mode=*/HIPDNN_CROSS_CORRELATION,
            /*computeType=*/HIPDNN_DATA_FLOAT));

    int new_batch_size, new_channels, new_height, new_width;
    checkCUDNN(hipdnnGetConvolution2dForwardOutputDim(convolution_descriptor,
                                                     input_descriptor,
                                                     kernel_descriptor,
                                                     &new_batch_size,
                                                     &new_channels,
                                                     &new_height,
                                                     &new_width));

    hipdnnTensorDescriptor_t output_descriptor;
    checkCUDNN(hipdnnCreateTensorDescriptor(&output_descriptor));
    checkCUDNN(hipdnnSetTensor4dDescriptor(output_descriptor,
            /*format=*/HIPDNN_TENSOR_NCHW,
            /*dataType=*/HIPDNN_DATA_FLOAT,
            /*batch_size=*/batch_size,
            /*channels=*/kernel_size,
            /*image_height=*/new_height,
            /*image_width=*/new_width));

    hipdnnConvolutionFwdAlgoPerf_t *convolution_algorithm = (hipdnnConvolutionFwdAlgoPerf_t*)malloc(sizeof(hipdnnConvolutionFwdAlgoPerf_t));
    int returned_algo_count = 0;
    checkCUDNN(
            cudnnGetConvolutionForwardAlgorithm_v7(cudnn,
                                                input_descriptor,
                                                kernel_descriptor,
                                                convolution_descriptor,
                                                output_descriptor,
                                                1,
                                                &returned_algo_count,
                                                convolution_algorithm));

    size_t workspace_bytes;
    checkCUDNN(hipdnnGetConvolutionForwardWorkspaceSize(cudnn,
                                                       input_descriptor,
                                                       kernel_descriptor,
                                                       convolution_descriptor,
                                                       output_descriptor,
                                                       convolution_algorithm[0].algo,
                                                       &workspace_bytes));

    void* d_workspace = NULL;
    if (workspace_bytes > 0) {
        hipMalloc(&d_workspace, workspace_bytes);
    }

    float *d_input = input;

    int image_bytes = batch_size * new_channels * new_height * new_width * sizeof(float);
    float* d_output{nullptr};
    hipMalloc(&d_output, image_bytes);
    hipMemset(d_output, 0, image_bytes);

    // Mystery kernel
    const float kernel_template[3][3] = {
            {0, -1, 0},
            {-1, 4, -1},
            {0,  -1, 0}
    };

    float h_kernel[kernel_size][3][3][3];
    for (int kernel = 0; kernel < kernel_size; ++kernel) {
        for (int channel = 0; channel < 3; ++channel) {
            for (int row = 0; row < 3; ++row) {
                for (int column = 0; column < 3; ++column) {
                    h_kernel[kernel][channel][row][column] = kernel_template[row][column];
                }
            }
        }
    }


    float* d_kernel{nullptr};
    hipMalloc(&d_kernel, sizeof(h_kernel));
    hipMemcpy(d_kernel, h_kernel, sizeof(h_kernel), hipMemcpyHostToDevice);

    const float alpha = 1, beta = 0;
    checkCUDNN(hipdnnConvolutionForward(cudnn,
                                       &alpha,
                                       input_descriptor,
                                       d_input,
                                       kernel_descriptor,
                                       d_kernel,
                                       convolution_descriptor,
                                       convolution_algorithm[0].algo,
                                       d_workspace,
                                       workspace_bytes,
                                       &beta,
                                       output_descriptor,
                                       d_output));
    output_shape[0] = batch_size;
    output_shape[1] = new_height;
    output_shape[2] = new_width;
    output_shape[3] = new_channels;
    return d_output;
}


float*
cuda_dnn_conv2d_float32_backward(float *data_output,
                                 float *data_input,
                                 float *data_filter,
                                 float alpha,
                                 float beta,
                                 int batch_size,
                                 int height,
                                 int width,
                                 int num_channels,
                                 int kernel_size,
                                 char padding)
{
    hipdnnHandle_t cudnn;
    hipdnnCreate(&cudnn);

    hipdnnConvolutionBwdDataAlgo_t algo = HIPDNN_CONVOLUTION_BWD_DATA_ALGO_1;

    void *workSpace = 0;
    size_t workSpaceSize;

    hipdnnTensorDescriptor_t input_descriptor;
    checkCUDNN(hipdnnCreateTensorDescriptor(&input_descriptor));
    checkCUDNN(hipdnnSetTensor4dDescriptor(input_descriptor,
            /*format=*/HIPDNN_TENSOR_NHWC,
            /*dataType=*/HIPDNN_DATA_FLOAT,
            /*batch_size=*/batch_size,
            /*channels=*/num_channels,
            /*image_height=*/height,
            /*image_width=*/width));

    hipdnnFilterDescriptor_t kernel_descriptor;
    checkCUDNN(hipdnnCreateFilterDescriptor(&kernel_descriptor));
    checkCUDNN(hipdnnSetFilter4dDescriptor(kernel_descriptor,
            /*dataType=*/HIPDNN_DATA_FLOAT,
            /*format=*/HIPDNN_TENSOR_NCHW,
            /*out_channels=*/kernel_size,
            /*in_channels=*/num_channels,
            /*kernel_height=*/3,
            /*kernel_width=*/3));

    int pad_height;
    int pad_width;
    switch (padding) {
        case 'v':
            pad_height = 0;
            pad_width = 0;
            break;
        case 's':
            pad_height = 1;
            pad_width = 1;
            break;
        default:
            pad_height = 0;
            pad_width = 0;
    }

    hipdnnConvolutionDescriptor_t convolution_descriptor;
    checkCUDNN(hipdnnCreateConvolutionDescriptor(&convolution_descriptor));
    checkCUDNN(hipdnnSetConvolution2dDescriptor(convolution_descriptor,
            /*pad_height=*/pad_height,
            /*pad_width=*/pad_width,
            /*vertical_stride=*/1,
            /*horizontal_stride=*/1,
            /*dilation_height=*/1,
            /*dilation_width=*/1,
            /*mode=*/HIPDNN_CONVOLUTION,
            /*computeType=*/HIPDNN_DATA_FLOAT));

    int new_batch_size, new_channels, new_height, new_width;
    checkCUDNN(hipdnnGetConvolution2dForwardOutputDim(convolution_descriptor,
                                                     input_descriptor,
                                                     kernel_descriptor,
                                                     &new_batch_size,
                                                     &new_channels,
                                                     &new_height,
                                                     &new_width));

    hipdnnTensorDescriptor_t output_descriptor;
    checkCUDNN(hipdnnCreateTensorDescriptor(&output_descriptor));
    checkCUDNN(hipdnnSetTensor4dDescriptor(output_descriptor,
            /*format=*/HIPDNN_TENSOR_NCHW,
            /*dataType=*/HIPDNN_DATA_FLOAT,
            /*batch_size=*/batch_size,
            /*channels=*/kernel_size,
            /*image_height=*/new_height,
            /*image_width=*/new_width));

    checkCUDNN ( hipdnnGetConvolutionBackwardDataWorkspaceSize(cudnn, kernel_descriptor, output_descriptor, convolution_descriptor,
                                                              input_descriptor, algo, &workSpaceSize) );

    if (workSpaceSize > 0) {
        hipMalloc(&workSpace, workSpaceSize);
    }
    checkCUDNN ( hipdnnConvolutionBackwardData (cudnn,
                                                  (void*)(&alpha),
                                                  kernel_descriptor, data_filter,
                                                  output_descriptor, data_output,
                                                  convolution_descriptor,
                                                  algo,
                                                  workSpace, workSpaceSize,
                                                  (void*)(&beta),
                                                  input_descriptor, data_input) );
    if (workSpace) {
        hipFree(workSpace);
        workSpace = 0;
    }
    return data_input;
}

#endif